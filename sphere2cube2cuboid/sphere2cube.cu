#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010

#include <vector>
#include <iostream>
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <opencv2/opencv.hpp>
#include <stdlib.h>


//////////////////////////////////////////////////
// Utils
//////////////////////////////////////////////////

#define PI2 1.5707963f
#define PI  3.1415927f
#define TAU 6.2831853f

#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32

__device__
float myLerp(float a, float b, float k)
{
    //return unsigned char(a * (1.f - k) + b * k);
    return a * (1.f - k) + b * k;
}

__device__
float myClamp(float value, float min, float max)
{
    if      (value < min) return min;
    else if (value > max) return max;
    else                  return value;
}

__device__
void normalize(float *v)
{
    const float k = 1.f / sqrtf(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);
    v[0] *= k;
    v[1] *= k;
    v[2] *= k;
}


//////////////////////////////////////////////////
// _to_img
//////////////////////////////////////////////////

__device__
void rect_to_img(int h, int w, const float* v, float& i, float& j)
{
    i = h * (       acosf (v[1])        / PI);
    j = w * (0.5f + atan2f(v[0], -v[2]) / TAU);
}

//////////////////////////////////////////////////
// _to_env
//////////////////////////////////////////////////

__device__
void cube_to_env(unsigned f, float i, float j, int h, int w, float* v)
{
    const int p[6][3][3] = {
        {{  0,  0, -1 }, {  0, -1,  0 }, {  1,  0,  0 }},
        {{  0,  0,  1 }, {  0, -1,  0 }, { -1,  0,  0 }},
        {{  1,  0,  0 }, {  0,  0,  1 }, {  0,  1,  0 }},
        {{  1,  0,  0 }, {  0,  0, -1 }, {  0, -1,  0 }},
        {{  1,  0,  0 }, {  0, -1,  0 }, {  0,  0,  1 }},
        {{ -1,  0,  0 }, {  0, -1,  0 }, {  0,  0, -1 }},
    };

    const float y = 2.0f * i / h - 1.0f;
    const float x = 2.0f * j / w - 1.0f;

    v[0] = p[f][0][0] * x + p[f][1][0] * y + p[f][2][0];
    v[1] = p[f][0][1] * x + p[f][1][1] * y + p[f][2][1];
    v[2] = p[f][0][2] * x + p[f][1][2] * y + p[f][2][2];

    normalize(v);
}

//////////////////////////////////////////////////
// Filters
//////////////////////////////////////////////////

__device__
void filter_linear(const unsigned char* srcImg, int srcHeight, int srcWidth, float i, float j, float* p)
{
    const float ii = myClamp(i - 0.5f, 0.0f, srcHeight - 1.0f);
    const float jj = myClamp(j - 0.5f, 0.0f, srcWidth - 1.0f);

    const long  i0 = lrintf(floorf(ii)), i1 = lrintf(ceilf(ii));
    const long  j0 = lrintf(floorf(jj)), j1 = lrintf(ceilf(jj));

    const float di = ii - i0;
    const float dj = jj - j0;
    /* int x = blockDim.x*blockIdx.x + threadIdx.x; */
    /* int y = blockDim.y*blockIdx.y + threadIdx.y; */
    /* printf("[%d, %d] => [%d, %d, %d, %d]\n", x, y, i0, j0, i1, j1); */
    for(int k=0; k<3; k++) {
        p[k] += myLerp(
            myLerp(srcImg[i0*srcWidth*3 + j0*3 + k], srcImg[i0*srcWidth*3 + j1*3 + k], dj),
            myLerp(srcImg[i1*srcWidth*3 + j0*3 + k], srcImg[i1*srcWidth*3 + j1*3 + k], dj),
            di
            );
    }
}

//////////////////////////////////////////////////
// supersample
//////////////////////////////////////////////////

__global__
void cudaRoutine(
    const unsigned char *srcImg,
    unsigned char *dstImgs,
    const unsigned srcHeight,
    const unsigned srcWidth,
    const unsigned dstLen
)
{
    for(int f = 0; f < 6; f++) {
        int i = blockDim.x*blockIdx.x + threadIdx.x;
        int j = blockDim.y*blockIdx.y + threadIdx.y;

        // Pattern
        const int rgss_pattern_size = 4;
        const float rgss_pattern[4][2] = {
            { 0.125f, 0.625f },
            { 0.375f, 0.125f },
            { 0.625f, 0.875f },
            { 0.875f, 0.375f }
        };

        float p[3] = { 0.0f, 0.0f, 0.0f };
        float v[3] = { 0.0f, 0.0f, 0.0f };
        float I, J;

        for(int k=0 ; k<rgss_pattern_size ; k++) {
            const float ii = rgss_pattern[k][0] + i;
            const float jj = rgss_pattern[k][1] + j;

            cube_to_env(f, ii, jj, dstLen, dstLen, v);
            rect_to_img(srcHeight, srcWidth, v, I, J);
            filter_linear(srcImg, srcHeight, srcWidth, I, J, p);
        }
        for(int k=0 ; k<3 ; k++) {
            dstImgs[(f*dstLen*dstLen*3)+(i*dstLen+j)*3 + k] = p[k] / rgss_pattern_size;
        }
    }
}

void sphere2cube(unsigned char* srcImg, 
        std::vector<unsigned char*>& dstImgs, 
        const unsigned int srcHeight, 
        const unsigned int srcWidth, 
        const unsigned int dstLen)
{
    // 3 : channels, 6 : output has 6 images
    const unsigned int srcSize = 3 * srcHeight * srcWidth;
    const unsigned int dstSize = 3 * dstLen * dstLen;
    static unsigned char *dSrcImg = NULL;
    static unsigned char *dDstImgs = NULL;

    if(!dSrcImg) {
        hipMalloc( (void**)&dSrcImg, srcSize * sizeof(unsigned char));
        hipMalloc( (void**)&dDstImgs, 6 * dstSize * sizeof(unsigned char));
    }

    hipMemcpy(dSrcImg, srcImg, srcSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 dimGrid(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 dimBlock(dstLen/BLOCK_DIM_X, dstLen/BLOCK_DIM_Y);

    cudaRoutine<<<dimGrid, dimBlock>>>(dSrcImg, dDstImgs, srcHeight, srcWidth, dstLen);

    for(int i = 0 ; i < 6; i++) {
        hipMemcpy(dstImgs[i], &dDstImgs[i*dstSize], dstSize, hipMemcpyDeviceToHost);
    }
}
